#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#define TILE_SX 32
#define TILE_SY 32

double __device__ dx(const double * v) {
    return 0.5*(v[+1] - v[-1]);
}

double __device__ dy(const double * v, const int dj) {
    return 0.5*(v[dj] - v[-dj]);
}

#define CUDA_CHECK do { \
  hipError_t res = hipGetLastError(); \
  if(res != hipSuccess) { \
    std::cerr << "CUDA Failure at " << __LINE__ << " " << hipGetErrorString(res) << "\n"; \
    exit(1); \
  } \
} while(0) 

void __global__ derivs(const double * v, double *v_x, double *v_y, double *v_xy) {
  double __shared__ tile[TILE_SX * TILE_SY];
  double __shared__ tile_x[TILE_SX * TILE_SY];

  int idx = threadIdx.x + threadIdx.y*TILE_SX;
  tile[idx] = v[idx]; 
  __syncthreads();
  
  if(threadIdx.x > 0 && threadIdx.x < TILE_SX - 1) {
    v_x[idx] = tile_x[idx] = dx(&tile[idx]);
  }

  __syncthreads();

  if(threadIdx.y > 0 && threadIdx.y < TILE_SX - 1) {
    v_y[idx] = dy(&tile[idx], TILE_SY);
    v_xy[idx] = dy(&tile_x[idx], TILE_SY);
  }
}

double now() {
  timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + t.tv_usec*1e-6;
}

int main(void) {
  double *v = (double*)calloc(TILE_SX * TILE_SY, sizeof(*v));
  double *v_x = (double*)calloc(TILE_SX * TILE_SY, sizeof(*v_x));
  double *v_y = (double*)calloc(TILE_SX * TILE_SY, sizeof(*v_y));
  double *v_xy = (double*)calloc(TILE_SX * TILE_SY, sizeof(*v_xy));
  double *d_v, *d_v_x, *d_v_y, *d_v_xy;
  hipMalloc(&d_v, TILE_SX * TILE_SY * sizeof(*v));
  CUDA_CHECK;
  hipMalloc(&d_v_x, TILE_SX * TILE_SY * sizeof(*v_x));
  CUDA_CHECK;
  hipMalloc(&d_v_y, TILE_SX * TILE_SY * sizeof(*v_y));
  CUDA_CHECK;
  hipMalloc(&d_v_xy, TILE_SX * TILE_SY * sizeof(*v_xy));
  CUDA_CHECK;

  double start = now();
  // TODO: run one without timint to get rid of initialization ost
  for(int i = 0 ; i < 1000 ; i++) {
    hipMemcpy(d_v, v, TILE_SX * TILE_SY * sizeof(*v), hipMemcpyHostToDevice);
    CUDA_CHECK;
    dim3 dimBlock(TILE_SX, TILE_SY);
    dim3 dimGrid(1, 1);
    derivs<<<dimGrid, dimBlock>>>(d_v, d_v_x, d_v_y, d_v_xy);
    CUDA_CHECK;
    hipMemcpy(v_x, d_v_x, TILE_SX * TILE_SY * sizeof(*v_x), hipMemcpyDeviceToHost);
    CUDA_CHECK;
    hipMemcpy(v_y, d_v_y, TILE_SX * TILE_SY * sizeof(*v_y), hipMemcpyDeviceToHost);
    CUDA_CHECK;
    hipMemcpy(v_xy, d_v_xy, TILE_SX * TILE_SY * sizeof(*v_xy), hipMemcpyDeviceToHost);
    CUDA_CHECK;
  }
  double end = now();
  std::cout << "took " << (end-start) << " s\n";

  return 0;
}
